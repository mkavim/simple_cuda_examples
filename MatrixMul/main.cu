/**
* 两个矩阵相乘：
* C = A * B
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define W 512
#define H 512
#define TB 32 

__global__ void MatrixMulKernel(float *C, float *A, float *B, int width)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if ((col >= width) || (row >= width)) return; // avoid illegal access
  float c = 0.0f;
  for (int k = 0; k < width; ++k){
      c += A[row * width + k] * B[k * width + col];
  }
  C[row * width + col] = c;
}

void display(float *Arr, int size){
    printf("size %d\n", size);
    for (int i=0; i < size ; ++i){
        printf("%f\t", Arr[i]);
        if (i % W == 0) printf("\n");
    }
    printf("\n");
}

int main()
{
  int mtsize = W * H;
  int msize = mtsize * sizeof(float);
  
  // 1、在Host端分配内存
  float *A = (float *)malloc(msize);
  float *B = (float *)malloc(msize);
  float *C = (float *)malloc(msize);
  
  // 2、初始化A和B
  for (int i = 0; i < mtsize; ++i)
  {
        A[i] = 1.0f;
        B[i] = 2.0f;
  } 

  // 3、在Device端为A和B分配内存
  float *d_A = NULL; 
  float *d_B = NULL; 
  hipMalloc(&d_A, msize);
  hipMalloc(&d_B, msize);
 
  // 4、将A和B从host端拷贝到Device端
  hipMemcpy(d_A, A, msize, hipMemcpyHostToDevice); 
  hipMemcpy(d_B, B, msize, hipMemcpyHostToDevice); 
  
  // 5、在Device端为C分配内存
  float *d_C = NULL; 
  hipMalloc(&d_C, msize);
  
  // 6、启动kernel计算
  dim3 dimBlock(TB, TB);
  int tb = (W + TB - 1) / TB;
  dim3 dimGrid(tb, tb);
  MatrixMulKernel<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, W);

  // 7、将计算结果从Device端复制回Host端
  hipMemcpy(C, d_C, msize, hipMemcpyDeviceToHost);
  display(C, mtsize);

  // 8、释放内存
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(A);
  free(B);
  free(C);
  
  return 0;
}
