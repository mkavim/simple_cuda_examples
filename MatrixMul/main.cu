/**
* 两个矩阵相乘：
* C = A * B
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define W 512
#define H 512
#define TB 32 

__global__ void MatrixMulKernel(float *C, float *A, float *B, int width)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if ((col >= width) || (row >= width)) return;
  float c = 0.0f;
      for (int k = 0; k < width; ++k){
         c += A[row * width + k] * B[k * width + col];
      }
      C[row*width + col] = c;
}

void display(float *Arr, int size){
    printf("size %d\n", size);
    for (int i=0; i < size ; ++i){
        printf("%f\t", Arr[i]);
        if (i % W == 0) printf("\n");
    }
    printf("\n");

}

int main()
{
  int mtsize = W*H;
  int msize = mtsize *sizeof(float);
  float *A = (float *)malloc(msize);
  float *B = (float *)malloc(msize);
  float *C = (float *)malloc(msize);
 
  for (int i = 0; i < mtsize; ++i)
  {
        A[i] = 1.0f;
        B[i] = 2.0f;
  } 
  printf("display C:...");
  //display(A, mtsize);
  //display(B, mtsize);
  //display(C, mtsize);

  // initial verctor a and b in cuda
  float *d_A = NULL; 
  float *d_B = NULL; 
  hipMalloc(&d_A, msize);
  hipMalloc(&d_B, msize);
 
  // copy to A,B to cuda
  hipMemcpy(d_A, A, msize, hipMemcpyHostToDevice); 
  hipMemcpy(d_B, B, msize, hipMemcpyHostToDevice); 
  
  float *d_C=NULL; 
  hipMalloc(&d_C, msize);
  
  dim3 dimBlock(TB, TB);
  int tb = (W+TB-1)/TB;
  dim3 dimGrid(tb, tb);

  MatrixMulKernel<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, W);

  // Copy results from device to host.
  hipMemcpy(C, d_C, msize, hipMemcpyDeviceToHost);
  display(C, mtsize);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(A);
  free(B);
  free(C);
  return 0;
}
